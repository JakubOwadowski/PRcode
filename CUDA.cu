
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <time.h>
#include <fstream>

#define BLOCK 16
#define number long long int

using namespace std;

__global__ void parallelTranspose(number* A, number* B, number m, number n) {
    number i = blockIdx.y * blockDim.y + threadIdx.y;
    number j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m && j < n) {
        B[j * m + i] = A[i * n + j];
    }
}

number strToNumber(char* char_) {
    char* pEnd;
    return strtoll(char_, &pEnd, 0);
}

int main(int argc, char *argv[]) {
    
    if (argc == 0) {
        printf("Error: no arguments specified.\n");
        return EXIT_FAILURE;
    }

    ofstream output;
    output.open ("results.txt");
    number *A, *B, n, m, i, max, size, *d_A, *d_B;
    max = strToNumber(argv[1]);
    size = sizeof(number);
    if (max < 0) {
        printf("Error: negative number.\n");
        return EXIT_FAILURE;
    }
    srand ((unsigned)time(NULL));
    m = 1000;
    while (true) {
        if (m  > max) break;
        n = 1000;
        while (true) {
            if (n > max) break;
            std::cout << "m: " << m << " n: " << n << endl;
            output << "m: " << m << " n: " << n << endl;
            clock_t begin, end;
            double elapsed_secs;

            A = (number*) malloc(m * n * size);
            B = (number*) malloc(m * n * size);

            for (i = 0; i < m * n; ++i)
                A[i] = rand() % 10;

            hipMalloc(&d_A, n * m * size);
            hipMalloc(&d_B, n * m * size);

            //dimensions
            dim3 threadblock(BLOCK, BLOCK);
            dim3 grid(1 + n / threadblock.x, 1 + m / threadblock.y);

            //copying A to the GPU
            hipMemcpy(d_A, A, n * m * size, hipMemcpyHostToDevice);

            //START PARALLEL
            elapsed_secs = 0;
            for (int i = 0; i < 10; i++) {
                begin = clock();

                //calling function
                parallelTranspose<<<grid, threadblock>>>(d_A, d_B, m, n);
                hipDeviceSynchronize();

                //once the function has been called I copy the result in matrix
                hipMemcpy(B, d_B, n * m * size, hipMemcpyDeviceToHost);

                end = clock();
                elapsed_secs += double(end - begin) / CLOCKS_PER_SEC;
                std::cout << ".";
            }
            std::cout << elapsed_secs/10 << std::endl;
            output << elapsed_secs/10 << std::endl;
            //END PARALLEL

            hipFree(d_A);
            hipFree(d_B);

            free(A);
            free(B);
            output << endl;
            n += 1000;
        } 
        m += 1000;
    }
    return 0;
}
